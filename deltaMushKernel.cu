

#include "hip/hip_runtime.h"

#include <iostream>
#include <cstdio>
__global__ void push_kernel(float * d_in_buffer, float * d_out_buffer, 
                            int * d_neighbours, const int size, int iter)
{

    int s_id = ((blockDim.x * blockIdx.x) +threadIdx.x)*3;
    int d_id =  ((blockDim.x * blockIdx.x) +threadIdx.x)*4;
    /*
    if (id>500 && id<600)
    {
        printf("%i \n",id); 
    }
    if (id==552)
    {
        printf("%f %f %f",d_in_buffer[id],d_in_buffer[id+1],d_in_buffer[id+2]);
    }
    
    */
    float * src;
    float * trg;
    for (int it=0; it<iter; it++)
    {}
    if(s_id<(size)*3)
    {
       
        int id;
        float v[3] = {0.0f,0.0f,0.0f};
        for (int i=0; i<4;i++)
        {
            id = d_neighbours[d_id+i]*3;
            v[0] += d_in_buffer[id]; 
            v[1] += d_in_buffer[id+1]; 
            v[2] += d_in_buffer[id+2]; 
        }
        v[0]/= 4.0f;
        v[1]/= 4.0f;
        v[2]/= 4.0f;
        d_out_buffer[s_id] = v[0]; 
        d_out_buffer[s_id+1] = v[1]; 
        d_out_buffer[s_id+2] = v[2]; 
    
    }
}


float * allocate_bufferFloat(int size, int stride)
{
    float * buffer;
    hipError_t result;
    result = hipMalloc((void **) &buffer,stride*size * sizeof(float));
    if (result != hipSuccess) 
            printf("Error: %s\n", hipGetErrorString(result));
    return buffer;
}
int * allocate_bufferInt(int size, int stride)
{
    int * buffer;
    hipError_t result;
    result = hipMalloc((void **) &buffer,stride*size * sizeof(int));
    if (result != hipSuccess) 
            printf("Error: %s\n", hipGetErrorString(result));
    return buffer;
}
void kernel_tear_down(float * d_in_buffer, float * d_out_buffer)
{
    if(d_in_buffer);
    {
        hipFree(d_in_buffer);
        d_in_buffer =0;
    }

    if(d_out_buffer)
    {
        hipFree(d_out_buffer);
        d_out_buffer=0;
    }
}

void average_launcher(const float * h_in_buffer, float * h_out_buffer, 
                   float * d_in_buffer, float * d_out_buffer, 
                   int * h_neighbours, int* d_neighbours,
                   const int size,int iter)
{
    //copy the memory from cpu to gpu
    int buffer_size = 3*size*sizeof(float);
    
    hipError_t s = hipMemcpy(d_in_buffer, h_in_buffer, buffer_size, hipMemcpyHostToDevice);
    if (s != hipSuccess) 
        printf("Error copying : %s\n", hipGetErrorString(s));
    
    //std::cout<<(*h_neighbours)[10]<<std::endl;
    s = hipMemcpy(d_neighbours, h_neighbours, 4*size*sizeof(int), hipMemcpyHostToDevice);
    if (s != hipSuccess) 
        printf("Error copying neigh_table: %s\n", hipGetErrorString(s));
    
    //setup the kernel
    int grain_size =128;
    size_t width_blocks = ((size%grain_size) != 0)?(size/grain_size) +1: (size/grain_size); 
    dim3 block_size(grain_size,1,1);
    dim3 grid_size(width_blocks,1,1);
    
    float * src= d_in_buffer;
    float * trg = d_out_buffer; 
    float * tmp;
    for (int i =0; i<iter; i++)
    {
        push_kernel<<<grid_size, block_size>>>(src, trg, d_neighbours, size, iter);
        tmp = src;
       src = trg;
      trg =tmp; 
    }
        //cudaDeviceSynchronize();
    //copy data back
    s = hipMemcpy(h_out_buffer, d_out_buffer, 3*size*sizeof(float), hipMemcpyDeviceToHost);
    if (s != hipSuccess) 
            printf("Error copying back: %s\n", hipGetErrorString(s));
}
